#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <thread>
#include "apex_api.hpp"

#define RUNTIME_API_CALL(apiFuncCall)                                          \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
        exit(-1);                                                              \
    }                                                                          \
} while (0)

struct DataElement
{
  char *name;
  int value;
};

__global__
void Kernel(DataElement *elem) {
  printf("On device: name=%s, value=%d\n", elem->name, elem->value);

  elem->name[0] = 'd';
  elem->value++;
}

void launch(DataElement *elem) {
  //APEX_SCOPED_TIMER;
  Kernel<<< 1, 1 >>>(elem);
  RUNTIME_API_CALL(hipDeviceSynchronize());
}

int main(int argc, char * argv[])
{
  APEX_UNUSED(argc);
  APEX_UNUSED(argv);
  apex::init("apex::cuda unit test", 0, 1);
  apex::apex_options::use_screen_output(true);
  DataElement *e;
  RUNTIME_API_CALL(hipMallocManaged((void**)&e, sizeof(DataElement)));
  RUNTIME_API_CALL(hipMallocManaged((void**)&(e->name), sizeof(char) * (strlen("hello") + 1) ));
  e->value = 10;
  strcpy(e->name, "hello");

  std::vector<std::thread*> threads;

  unsigned i;
  //unsigned test_numthreads = apex::hardware_concurrency() - 1;
  unsigned test_numthreads = 3;
  for(i = 0 ; i < test_numthreads ; i++) {
    std::thread * worker = new std::thread(launch,e);
    threads.push_back(worker);
  }
  launch(e);
  for(i = 0 ; i < test_numthreads ; i++) {
    threads[i]->join();
  }

  printf("On host: name=%s, value=%d\n", e->name, e->value);

  RUNTIME_API_CALL(hipFree(e->name));
  RUNTIME_API_CALL(hipFree(e));
  apex::finalize();
  apex::cleanup();
}
